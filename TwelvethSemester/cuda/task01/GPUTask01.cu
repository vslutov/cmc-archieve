#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <stdexcept>
#include <cstring>
#include <memory>
#include <ctime>

#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <SOIL/SOIL.h>

const float EPS = 0.15 * 0.15;
const ssize_t BOX_SIZE = 5;
const ssize_t PAD_SIZE = BOX_SIZE / 2;
const ssize_t BLOCK_SIZE_X = 16;
const ssize_t BLOCK_SIZE_Y = 16;
const dim3 BLOCK_SIZE = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y);
const ssize_t SHARED_BUFFER_SIZE_X = BLOCK_SIZE_X + BOX_SIZE - 1;
const ssize_t SHARED_BUFFER_SIZE_Y = BLOCK_SIZE_Y + BOX_SIZE - 1;

// Access to array
#define ACCESS(arr, x, y, c, w, h, channels) (arr[((y) * w + (x)) * (channels) + (c)])

// Check that point is outside of array
__host__ __device__ int
validAddress(int x, int y, int w, int h)
{
  return x >= 0 && x < w && y >= 0 && y < h;
}

inline void
calcTmp (
  uint8_t *inImg,
  float *tmpImg,
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  for (ssize_t c = 0; c < 3; ++c) {
    float mean = 0;
    float corr = 0;
    ssize_t count = 0;

    for (ssize_t i = x - PAD_SIZE; i < x + BOX_SIZE - PAD_SIZE; ++ i) {
      for (ssize_t j = y - PAD_SIZE; j < y + BOX_SIZE - PAD_SIZE; ++ j) {
        if (validAddress(i, j, w, h)) {
          float color = ACCESS(inImg, i, j, c, w, h, 3) / 255.0;
          mean += color;
          corr += color * color;
          ++ count;
        }
      }
    }

    mean /= count;
    corr /= count;
    float var = corr - mean * mean;

    float a = var / (var + eps);
    float b = (1 - a) * mean;

    ACCESS(tmpImg, x, y, 2 * c, w, h, 6) = a;
    ACCESS(tmpImg, x, y, 2 * c + 1, w, h, 6) = b;
  }
}

__device__ void
prepareSharedBufferFromIn(
  uint8_t *inImg,
  float sharedBuffer[SHARED_BUFFER_SIZE_X][SHARED_BUFFER_SIZE_Y][3],
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h
)
{
  ssize_t diffX = -PAD_SIZE;
  ssize_t diffY = -PAD_SIZE;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 3; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(inImg, x + diffX, y + diffY, c, w, h, 3) / 255.0;
    }
  }
  diffY = BOX_SIZE - PAD_SIZE - 1;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 3; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(inImg, x + diffX, y + diffY, c, w, h, 3) / 255.0;
    }
  }
  diffX = BOX_SIZE - PAD_SIZE - 1;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 3; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(inImg, x + diffX, y + diffY, c, w, h, 3) / 255.0;
    }
  }
  diffY = -PAD_SIZE;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 3; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(inImg, x + diffX, y + diffY, c, w, h, 3) / 255.0;
    }
  }
  __syncthreads();
}

__device__  void
calcTmpShared(
  uint8_t *inImg,
  float *tmpImg,
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  __shared__ float sharedBuffer[SHARED_BUFFER_SIZE_X][SHARED_BUFFER_SIZE_Y][3];
  prepareSharedBufferFromIn(inImg, sharedBuffer, x, y, w, h);

  for (ssize_t c = 0; c < 3; ++c) {
    float mean = 0;
    float corr = 0;
    ssize_t count = 0;

    for (ssize_t i = x - PAD_SIZE; i < x + BOX_SIZE - PAD_SIZE; ++ i) {
      for (ssize_t j = y - PAD_SIZE; j < y + BOX_SIZE - PAD_SIZE; ++ j) {
        if (validAddress(i, j, w, h)) {
          float color = sharedBuffer[i - blockIdx.x * blockDim.x + PAD_SIZE][j - blockIdx.y * blockDim.y + PAD_SIZE][c];
          mean += color;
          corr += color * color;
          ++ count;
        }
      }
    }

    mean /= count;
    corr /= count;
    float var = corr - mean * mean;

    float a = var / (var + eps);
    float b = (1 - a) * mean;

    ACCESS(tmpImg, x, y, 2 * c, w, h, 6) = a;
    ACCESS(tmpImg, x, y, 2 * c + 1, w, h, 6) = b;
  }
}

__device__ void
prepareSharedBufferFromTmp(
  float *tmpImg,
  float sharedBuffer[SHARED_BUFFER_SIZE_X][SHARED_BUFFER_SIZE_Y][6],
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h
)
{
  ssize_t diffX = -PAD_SIZE;
  ssize_t diffY = -PAD_SIZE;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 6; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(tmpImg, x + diffX, y + diffY, c, w, h, 6);
    }
  }
  diffY = BOX_SIZE - PAD_SIZE - 1;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 6; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(tmpImg, x + diffX, y + diffY, c, w, h, 6);
    }
  }
  diffX = BOX_SIZE - PAD_SIZE - 1;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 6; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(tmpImg, x + diffX, y + diffY, c, w, h, 6);
    }
  }
  diffY = -PAD_SIZE;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 6; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(tmpImg, x + diffX, y + diffY, c, w, h, 6);
    }
  }
  __syncthreads();
}

inline  void
calcOut(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h
)
{
  for (ssize_t c = 0; c < 3; ++c) {
    float meanA = 0;
    float meanB = 0;
    ssize_t count = 0;

    for (ssize_t i = x - PAD_SIZE; i < x + BOX_SIZE - PAD_SIZE; ++ i) {
      for (ssize_t j = y - PAD_SIZE; j < y + BOX_SIZE - PAD_SIZE; ++ j) {
        if (validAddress(i, j, w, h)) {
          meanA += ACCESS(tmpImg, i, j, 2 * c, w, h, 6);
          meanB += ACCESS(tmpImg, i, j, 2 * c + 1, w, h, 6);
          ++ count;
        }
      }
    }

    meanA /= count;
    meanB /= count;

    ACCESS(outImg, x, y, c, w, h, 3) = ACCESS(inImg, x, y, c, w, h, 3) * meanA + 255 * meanB;
  }
}

__device__  void
calcOutShared(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h
)
{
  __shared__ float sharedBuffer[SHARED_BUFFER_SIZE_X][SHARED_BUFFER_SIZE_Y][6];
  prepareSharedBufferFromTmp(tmpImg, sharedBuffer, x, y, w, h);

  for (ssize_t c = 0; c < 3; ++c) {
    float meanA = 0;
    float meanB = 0;
    ssize_t count = 0;

    for (ssize_t i = x - PAD_SIZE; i < x + BOX_SIZE - PAD_SIZE; ++ i) {
      for (ssize_t j = y - PAD_SIZE; j < y + BOX_SIZE - PAD_SIZE; ++ j) {
        if (validAddress(i, j, w, h)) {
          meanA += sharedBuffer[i - blockIdx.x * blockDim.x + PAD_SIZE][j - blockIdx.y * blockDim.y + PAD_SIZE][2 * c];
          meanB += sharedBuffer[i - blockIdx.x * blockDim.x + PAD_SIZE][j - blockIdx.y * blockDim.y + PAD_SIZE][2 * c + 1];
          ++ count;
        }
      }
    }

    meanA /= count;
    meanB /= count;

    ACCESS(outImg, x, y, c, w, h, 3) = ACCESS(inImg, x, y, c, w, h, 3) * meanA + 255 * meanB;
  }
}

void
guidedCpu(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  for(ssize_t x = 0; x < w; ++x) {
    for(ssize_t y = 0; y < h; ++y) {
      calcTmp(inImg, tmpImg, x, y, w, h, eps);
    }
  }
  for(ssize_t x = 0; x < w; ++x) {
    for(ssize_t y = 0; y < h; ++y) {
      calcOut(inImg, tmpImg, outImg, x, y, w, h);
    }
  }
}

void
guidedOpenMP(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  #pragma omp parallel for
  for(ssize_t x = 0; x < w; x++) {
    for(ssize_t y = 0; y < h; y++) {
      calcTmp(inImg, tmpImg, x, y, w, h, eps);
    }
  }
  #pragma omp parallel for
  for(ssize_t x = 0; x < w; x++) {
    for(ssize_t y = 0; y < h; y++) {
      calcOut(inImg, tmpImg, outImg, x, y, w, h);
    }
  }
}

__global__ void
guidedGpuTmp(
  uint8_t *inImg,
  float *tmpImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  // compute idx
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (!validAddress(x, y, w, h)) {
    return;
  }

  calcTmpShared(inImg, tmpImg, x, y, w, h, eps);
}

__global__ void
guidedGpuOut(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  // compute idx
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (!validAddress(x, y, w, h)) {
    return;
  }

  calcOutShared(inImg, tmpImg, outImg, x, y, w, h);
}

void
checkErrors()
{
  hipError_t code = hipGetLastError();
  if (code != hipSuccess){
    throw std::runtime_error(std::string("Cuda error : ") + hipGetErrorString(code));
  }
}

void
guidedGpu(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  dim3 gridSize(std::ceil((float) w / BLOCK_SIZE_X),
                std::ceil((float) h / BLOCK_SIZE_Y));

  guidedGpuTmp<<<gridSize, BLOCK_SIZE>>>(inImg, tmpImg, w, h, eps); checkErrors();
  guidedGpuOut<<<gridSize, BLOCK_SIZE>>>(inImg, tmpImg, outImg, w, h, eps); checkErrors();
}

float
measureTime(const std::string &task, const std::function<void(void)> &func) {
  volatile std::clock_t start, time;

  start = std::clock();
  func();
  time = std::clock() - start;

  float result = time * 1000.0 / CLOCKS_PER_SEC;

  std::cout << task <<" time: " << result << " ms" << std::endl;

  return result;
}

int
main(int argc, char **argv)
{
  // input image
  if (argc < 2) {
    std::cerr << "Usage: " << argv[0] << " image" << std::endl;
    return 1;
  }

  int width, height;

  uint8_t* inImg = SOIL_load_image(
    argv[1],
    &width,
    &height,
    NULL,
    SOIL_LOAD_RGB
  );

  int image_size = height * width * 3;

  // output image
  auto outImgContainer = std::unique_ptr<uint8_t[]>(new uint8_t[image_size]);
  auto outImg = outImgContainer.get();
  auto tmpImgContainer = std::unique_ptr<float[]>(new float[2 * image_size]);
  auto tmpImg = tmpImgContainer.get();

  // guided arguments
  // cpu
  measureTime("CPU", [inImg, tmpImg, outImg, width, height]() {
    guidedCpu(inImg, tmpImg, outImg, width, height, EPS);
  });

  SOIL_save_image("cpu.tga", SOIL_SAVE_TYPE_TGA, width, height, 3, outImg);

  // OpenMP
  measureTime("OpenMP", [inImg, tmpImg, outImg, width, height]() {
    guidedOpenMP(inImg, tmpImg, outImg, width, height, EPS);
  });

  SOIL_save_image("openmp.tga", SOIL_SAVE_TYPE_TGA, width, height, 3, outImg);

  // allocate memory
  uint8_t *inImgDev, *outImgDev;
  float *tmpImgDev;
  hipMalloc((void **) &inImgDev, image_size); checkErrors();
  hipMalloc((void **) &outImgDev, image_size); checkErrors();
  hipMalloc((void **) &tmpImgDev, sizeof(float) * 2 * image_size); checkErrors();
  hipMemcpy(inImgDev, inImg, image_size, hipMemcpyHostToDevice); checkErrors();

  // cuda
  const auto timeWithCopy = measureTime("CUDA with copy", [inImgDev, tmpImgDev, outImgDev, width, height, inImg, outImg, image_size]() {
    hipMemcpy(inImgDev, inImg, image_size, hipMemcpyHostToDevice); checkErrors();
    guidedGpu(inImgDev, tmpImgDev, outImgDev, width, height, EPS);
    hipMemcpy(outImg, outImgDev, image_size, hipMemcpyDeviceToHost); checkErrors();
  });

  const auto timeWithoutCopy = measureTime("CUDA without copy", [inImgDev, tmpImgDev, outImgDev, width, height]() {
    guidedGpu(inImgDev, tmpImgDev, outImgDev, width, height, EPS);
    hipDeviceSynchronize(); checkErrors();
  });

  std::cout << "Copy time: " << (timeWithCopy - timeWithoutCopy) << " ms" << std::endl;

  SOIL_save_image(
    "gpu.tga",
    SOIL_SAVE_TYPE_TGA,
    width,
    height,
    3,
    outImg
  );

  hipFree(inImgDev); checkErrors();
  hipFree(outImgDev); checkErrors();

  return 0;
}

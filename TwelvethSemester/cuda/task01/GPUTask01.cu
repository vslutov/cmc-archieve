#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <stdexcept>
#include <cstring>
#include <memory>
#include <ctime>

#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <SOIL/SOIL.h>
#include "hip/hip_vector_types.h"

const float EPS = 0.15 * 0.15;
const ssize_t BOX_SIZE = 5;
const ssize_t PAD_SIZE = BOX_SIZE / 2;
const ssize_t BLOCK_SIZE_X = 16;
const ssize_t BLOCK_SIZE_Y = 16;
const dim3 BLOCK_SIZE = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y);
const ssize_t SHARED_BUFFER_SIZE_X = BLOCK_SIZE_X + BOX_SIZE - 1;
const ssize_t SHARED_BUFFER_SIZE_Y = BLOCK_SIZE_Y + BOX_SIZE - 1;

// Access to array
#define ACCESS(arr, x, y, c, w, h, channels) (arr[((y) * w + (x)) * (channels) + (c)])
#define ACCESS_TEXTURE(arr, x, y, w, h) tex2D(arr, float(x) / w, float(y) / h)

texture<float4, 2, hipReadModeElementType> tex;
texture<float4, 2, hipReadModeElementType> texA;
texture<float4, 2, hipReadModeElementType> texB;

// Check that point is outside of array
__host__ __device__ int
validAddress(int x, int y, int w, int h)
{
  return x >= 0 && x < w && y >= 0 && y < h;
}

__host__ __device__ inline void
calcTmp (
  uint8_t *inImg,
  float *tmpImg,
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  for (ssize_t c = 0; c < 3; ++c) {
    float mean = 0;
    float corr = 0;
    ssize_t count = 0;

    for (ssize_t i = x - PAD_SIZE; i < x + BOX_SIZE - PAD_SIZE; ++ i) {
      for (ssize_t j = y - PAD_SIZE; j < y + BOX_SIZE - PAD_SIZE; ++ j) {
        if (validAddress(i, j, w, h)) {
          float color = ACCESS(inImg, i, j, c, w, h, 3) / 255.0;
          mean += color;
          corr += color * color;
          ++ count;
        }
      }
    }

    mean /= count;
    corr /= count;
    float var = corr - mean * mean;

    float a = var / (var + eps);
    float b = (1 - a) * mean;

    ACCESS(tmpImg, x, y, 2 * c, w, h, 6) = a;
    ACCESS(tmpImg, x, y, 2 * c + 1, w, h, 6) = b;
  }
}

__device__ inline void
calcTmpTexture (
  float4 *tmpA,
  float4 *tmpB,
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  float4 mean = make_float4(0, 0, 0, 0);
  float4 corr = make_float4(0, 0, 0, 0);
  ssize_t count = 0;

  for (ssize_t i = x - PAD_SIZE; i < x + BOX_SIZE - PAD_SIZE; ++ i) {
    for (ssize_t j = y - PAD_SIZE; j < y + BOX_SIZE - PAD_SIZE; ++ j) {
      if (validAddress(i, j, w, h)) {
        float4 color = ACCESS_TEXTURE(tex, i, j, w, h) / 255.0;
        mean += color;
        corr += color * color;
        ++ count;
      }
    }
  }

  mean /= count;
  corr /= count;
  float4 var = corr - mean * mean;

  float4 a = var / (var + eps);
  float4 b = (make_float4(1, 1, 1, 1) - a) * mean;

  ACCESS(tmpA, x, y, 0, w, h, 1) = a;
  ACCESS(tmpB, x, y, 0, w, h, 1) = b;
}

__device__ void
prepareSharedBufferFromIn(
  uint8_t *inImg,
  float sharedBuffer[SHARED_BUFFER_SIZE_X][SHARED_BUFFER_SIZE_Y][3],
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h
)
{
  ssize_t diffX = -PAD_SIZE;
  ssize_t diffY = -PAD_SIZE;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 3; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(inImg, x + diffX, y + diffY, c, w, h, 3) / 255.0;
    }
  }
  diffY = BOX_SIZE - PAD_SIZE - 1;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 3; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(inImg, x + diffX, y + diffY, c, w, h, 3) / 255.0;
    }
  }
  diffX = BOX_SIZE - PAD_SIZE - 1;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 3; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(inImg, x + diffX, y + diffY, c, w, h, 3) / 255.0;
    }
  }
  diffY = -PAD_SIZE;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 3; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(inImg, x + diffX, y + diffY, c, w, h, 3) / 255.0;
    }
  }
  __syncthreads();
}

__device__  void
calcTmpShared(
  uint8_t *inImg,
  float *tmpImg,
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  __shared__ float sharedBuffer[SHARED_BUFFER_SIZE_X][SHARED_BUFFER_SIZE_Y][3];
  prepareSharedBufferFromIn(inImg, sharedBuffer, x, y, w, h);

  for (ssize_t c = 0; c < 3; ++c) {
    float mean = 0;
    float corr = 0;
    ssize_t count = 0;

    for (ssize_t i = x - PAD_SIZE; i < x + BOX_SIZE - PAD_SIZE; ++ i) {
      for (ssize_t j = y - PAD_SIZE; j < y + BOX_SIZE - PAD_SIZE; ++ j) {
        if (validAddress(i, j, w, h)) {
          float color = sharedBuffer[i - blockIdx.x * blockDim.x + PAD_SIZE][j - blockIdx.y * blockDim.y + PAD_SIZE][c];
          mean += color;
          corr += color * color;
          ++ count;
        }
      }
    }

    mean /= count;
    corr /= count;
    float var = corr - mean * mean;

    float a = var / (var + eps);
    float b = (1 - a) * mean;

    ACCESS(tmpImg, x, y, 2 * c, w, h, 6) = a;
    ACCESS(tmpImg, x, y, 2 * c + 1, w, h, 6) = b;
  }
}

__device__ void
prepareSharedBufferFromTmp(
  float *tmpImg,
  float sharedBuffer[SHARED_BUFFER_SIZE_X][SHARED_BUFFER_SIZE_Y][6],
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h
)
{
  ssize_t diffX = -PAD_SIZE;
  ssize_t diffY = -PAD_SIZE;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 6; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(tmpImg, x + diffX, y + diffY, c, w, h, 6);
    }
  }
  diffY = BOX_SIZE - PAD_SIZE - 1;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 6; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(tmpImg, x + diffX, y + diffY, c, w, h, 6);
    }
  }
  diffX = BOX_SIZE - PAD_SIZE - 1;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 6; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(tmpImg, x + diffX, y + diffY, c, w, h, 6);
    }
  }
  diffY = -PAD_SIZE;
  if (validAddress(x + diffX, y + diffY, w, h)) {
    for (ssize_t c = 0; c < 6; ++ c) {
      sharedBuffer[threadIdx.x + diffX + PAD_SIZE][threadIdx.y + diffY + PAD_SIZE][c] = ACCESS(tmpImg, x + diffX, y + diffY, c, w, h, 6);
    }
  }
  __syncthreads();
}

__host__ __device__ inline  void
calcOut(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h
)
{
  for (ssize_t c = 0; c < 3; ++c) {
    float meanA = 0;
    float meanB = 0;
    ssize_t count = 0;

    for (ssize_t i = x - PAD_SIZE; i < x + BOX_SIZE - PAD_SIZE; ++ i) {
      for (ssize_t j = y - PAD_SIZE; j < y + BOX_SIZE - PAD_SIZE; ++ j) {
        if (validAddress(i, j, w, h)) {
          meanA += ACCESS(tmpImg, i, j, 2 * c, w, h, 6);
          meanB += ACCESS(tmpImg, i, j, 2 * c + 1, w, h, 6);
          ++ count;
        }
      }
    }

    meanA /= count;
    meanB /= count;

    ACCESS(outImg, x, y, c, w, h, 3) = ACCESS(inImg, x, y, c, w, h, 3) * meanA + 255 * meanB;
  }
}

__device__ inline  void
calcOutTexture(
  uint8_t *outImg,
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h
)
{
  float4 meanA = make_float4(0, 0, 0, 0);
  float4 meanB = make_float4(0, 0, 0, 0);
  ssize_t count = 0;

  for (ssize_t i = x - PAD_SIZE; i < x + BOX_SIZE - PAD_SIZE; ++ i) {
    for (ssize_t j = y - PAD_SIZE; j < y + BOX_SIZE - PAD_SIZE; ++ j) {
      if (validAddress(i, j, w, h)) {
        meanA = meanA + ACCESS_TEXTURE(texA, i, j, w, h);
        meanB = meanB + ACCESS_TEXTURE(texB, i, j, w, h);
        ++ count;
      }
    }
  }

  meanA /= count;
  meanB /= count;

  float4 result = ACCESS_TEXTURE(tex, x, y, w, h) * meanA + 255 * meanB;
  ACCESS(outImg, x, y, 0, w, h, 3) = result.x;
  ACCESS(outImg, x, y, 1, w, h, 3) = result.y;
  ACCESS(outImg, x, y, 2, w, h, 3) = result.z;
}

__device__  void
calcOutShared(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t x,
  ssize_t y,
  ssize_t w,
  ssize_t h
)
{
  __shared__ float sharedBuffer[SHARED_BUFFER_SIZE_X][SHARED_BUFFER_SIZE_Y][6];
  prepareSharedBufferFromTmp(tmpImg, sharedBuffer, x, y, w, h);

  for (ssize_t c = 0; c < 3; ++c) {
    float meanA = 0;
    float meanB = 0;
    ssize_t count = 0;

    for (ssize_t i = x - PAD_SIZE; i < x + BOX_SIZE - PAD_SIZE; ++ i) {
      for (ssize_t j = y - PAD_SIZE; j < y + BOX_SIZE - PAD_SIZE; ++ j) {
        if (validAddress(i, j, w, h)) {
          meanA += sharedBuffer[i - blockIdx.x * blockDim.x + PAD_SIZE][j - blockIdx.y * blockDim.y + PAD_SIZE][2 * c];
          meanB += sharedBuffer[i - blockIdx.x * blockDim.x + PAD_SIZE][j - blockIdx.y * blockDim.y + PAD_SIZE][2 * c + 1];
          ++ count;
        }
      }
    }

    meanA /= count;
    meanB /= count;

    ACCESS(outImg, x, y, c, w, h, 3) = ACCESS(inImg, x, y, c, w, h, 3) * meanA + 255 * meanB;
  }
}

void
guidedCpu(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  for(ssize_t x = 0; x < w; ++x) {
    for(ssize_t y = 0; y < h; ++y) {
      calcTmp(inImg, tmpImg, x, y, w, h, eps);
    }
  }
  for(ssize_t x = 0; x < w; ++x) {
    for(ssize_t y = 0; y < h; ++y) {
      calcOut(inImg, tmpImg, outImg, x, y, w, h);
    }
  }
}

void
guidedOpenMP(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  #pragma omp parallel for
  for(ssize_t x = 0; x < w; x++) {
    for(ssize_t y = 0; y < h; y++) {
      calcTmp(inImg, tmpImg, x, y, w, h, eps);
    }
  }
  #pragma omp parallel for
  for(ssize_t x = 0; x < w; x++) {
    for(ssize_t y = 0; y < h; y++) {
      calcOut(inImg, tmpImg, outImg, x, y, w, h);
    }
  }
}

__global__ void
guidedGpuTmpGlobal(
  uint8_t *inImg,
  float *tmpImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  // compute idx
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (!validAddress(x, y, w, h)) {
    return;
  }

  calcTmp(inImg, tmpImg, x, y, w, h, eps);
}

__global__ void
guidedGpuTmpTexture(
  float4 *tmpA,
  float4 *tmpB,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  // compute idx
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (!validAddress(x, y, w, h)) {
    return;
  }

  calcTmpTexture(tmpA, tmpB, x, y, w, h, eps);
}

__global__ void
guidedGpuOutGlobal(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  // compute idx
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (!validAddress(x, y, w, h)) {
    return;
  }

  calcOut(inImg, tmpImg, outImg, x, y, w, h);
}

__global__ void
guidedGpuOutTexture(
  uint8_t *outImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  // compute idx
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (!validAddress(x, y, w, h)) {
    return;
  }

  calcOutTexture(outImg, x, y, w, h);
}

__global__ void
guidedGpuTmpShared(
  uint8_t *inImg,
  float *tmpImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  // compute idx
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (!validAddress(x, y, w, h)) {
    return;
  }

  calcTmpShared(inImg, tmpImg, x, y, w, h, eps);
}

__global__ void
guidedGpuOutShared(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  // compute idx
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (!validAddress(x, y, w, h)) {
    return;
  }

  calcOutShared(inImg, tmpImg, outImg, x, y, w, h);
}

void
checkErrors()
{
  hipError_t code = hipGetLastError();
  if (code != hipSuccess){
    throw std::runtime_error(std::string("Cuda error : ") + hipGetErrorString(code));
  }
}

void
guidedGpuShared(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  dim3 gridSize(std::ceil((float) w / BLOCK_SIZE_X),
                std::ceil((float) h / BLOCK_SIZE_Y));

  guidedGpuTmpShared<<<gridSize, BLOCK_SIZE>>>(inImg, tmpImg, w, h, eps); checkErrors();
  guidedGpuOutShared<<<gridSize, BLOCK_SIZE>>>(inImg, tmpImg, outImg, w, h, eps); checkErrors();
}

void
guidedGpuGlobal(
  uint8_t *inImg,
  float *tmpImg,
  uint8_t *outImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  dim3 gridSize(std::ceil((float) w / BLOCK_SIZE_X),
                std::ceil((float) h / BLOCK_SIZE_Y));

  guidedGpuTmpGlobal<<<gridSize, BLOCK_SIZE>>>(inImg, tmpImg, w, h, eps); checkErrors();
  guidedGpuOutGlobal<<<gridSize, BLOCK_SIZE>>>(inImg, tmpImg, outImg, w, h, eps); checkErrors();
}

void
guidedGpuTexture(
  float4 *tmpA,
  float4 *tmpB,
  hipArray *cuArrayA,
  hipArray *cuArrayB,
  uint8_t *outImg,
  ssize_t w,
  ssize_t h,
  float eps
)
{
  dim3 gridSize(std::ceil((float) w / BLOCK_SIZE_X),
                std::ceil((float) h / BLOCK_SIZE_Y));

  guidedGpuTmpTexture<<<gridSize, BLOCK_SIZE>>>(tmpA, tmpB, w, h, eps); checkErrors();

  hipMemcpyToArray(cuArrayA,
                    0,
                    0,
                    tmpA,
                    w * h * sizeof(float4),
                    hipMemcpyDeviceToDevice); checkErrors();

  hipMemcpyToArray(cuArrayB,
                    0,
                    0,
                    tmpB,
                    w * h * sizeof(float4),
                    hipMemcpyDeviceToDevice); checkErrors();

  guidedGpuOutTexture<<<gridSize, BLOCK_SIZE>>>(outImg, w, h, eps); checkErrors();
}

float
measureTime(const std::string &task, const std::function<void(void)> &func) {
  volatile std::clock_t start, time;

  start = std::clock();
  func();
  time = std::clock() - start;

  float result = time * 1000.0 / CLOCKS_PER_SEC;

  std::cout << task <<" time: " << result << " ms" << std::endl;

  return result;
}

int
main(int argc, char **argv)
{
  // input image
  if (argc < 2) {
    std::cerr << "Usage: " << argv[0] << " image" << std::endl;
    return 1;
  }

  int width, height;

  uint8_t* inImg = SOIL_load_image(
    argv[1],
    &width,
    &height,
    NULL,
    SOIL_LOAD_RGB
  );

  int image_size = height * width * 3;

  // output image
  auto outImgContainer = std::unique_ptr<uint8_t[]>(new uint8_t[image_size]);
  auto outImg = outImgContainer.get();
  auto tmpImgContainer = std::unique_ptr<float[]>(new float[2 * image_size]);
  auto tmpImg = tmpImgContainer.get();

  // guided arguments
  // cpu
  measureTime("CPU", [inImg, tmpImg, outImg, width, height]() {
    guidedCpu(inImg, tmpImg, outImg, width, height, EPS);
  });

  SOIL_save_image("cpu.tga", SOIL_SAVE_TYPE_TGA, width, height, 3, outImg);

  // OpenMP
  measureTime("OpenMP", [inImg, tmpImg, outImg, width, height]() {
    guidedOpenMP(inImg, tmpImg, outImg, width, height, EPS);
  });

  SOIL_save_image("openmp.tga", SOIL_SAVE_TYPE_TGA, width, height, 3, outImg);

  // allocate memory
  uint8_t *inImgDev, *outImgDev;
  float *tmpImgDev;
  hipMalloc((void **) &inImgDev, image_size); checkErrors();
  hipMalloc((void **) &outImgDev, image_size); checkErrors();
  hipMalloc((void **) &tmpImgDev, sizeof(float) * 2 * image_size); checkErrors();
  hipMemcpy(inImgDev, inImg, image_size, hipMemcpyHostToDevice); checkErrors();

  // Allocate array and copy image data
  auto hDataContainer = std::unique_ptr<float4[]>(new float4[width * height]);
  auto hData = hDataContainer.get();
  for (ssize_t i = 0; i < width * height; ++ i) {
    hData[i] = make_float4(inImg[i * 3], inImg[i * 3 + 1], inImg[i * 3 + 2], 0);
  }
  auto channelDesc = hipCreateChannelDesc<float4>();
  hipArray *cuArray;
  hipMallocArray(&cuArray,
                  &channelDesc,
                  width,
                  height); checkErrors();
  hipMemcpyToArray(cuArray,
                    0,
                    0,
                    hData,
                    width * height * sizeof(float4),
                    hipMemcpyHostToDevice); checkErrors();

  tex.addressMode[0] = hipAddressModeWrap;
  tex.addressMode[1] = hipAddressModeWrap;
  tex.filterMode = hipFilterModeLinear;
  tex.normalized = true;    // access with normalized texture coordinates
  hipBindTextureToArray(tex, cuArray, channelDesc); checkErrors();

  float4 *tmpA;
  hipMalloc((void **) &tmpA, sizeof(float4) * width * height); checkErrors();
  hipArray *cuArrayA;
  hipMallocArray(&cuArrayA,
                  &channelDesc,
                  width,
                  height); checkErrors();
  texA.addressMode[0] = hipAddressModeWrap;
  texA.addressMode[1] = hipAddressModeWrap;
  texA.filterMode = hipFilterModeLinear;
  texA.normalized = true;    // access with normalized texture coordinates
  hipBindTextureToArray(texA, cuArrayA, channelDesc); checkErrors();

  float4 *tmpB;
  hipMalloc((void **) &tmpB, sizeof(float4) * width * height); checkErrors();
  hipArray *cuArrayB;
  hipMallocArray(&cuArrayB,
                  &channelDesc,
                  width,
                  height); checkErrors();
  texB.addressMode[0] = hipAddressModeWrap;
  texB.addressMode[1] = hipAddressModeWrap;
  texB.filterMode = hipFilterModeLinear;
  texB.normalized = true;    // access with normalized texture coordinates
  hipBindTextureToArray(texB, cuArrayB, channelDesc); checkErrors();

  // cuda
  const auto timeWithCopy = measureTime("CUDA shared with copy", [inImgDev, tmpImgDev, outImgDev, width, height, inImg, outImg, image_size]() {
    hipMemcpy(inImgDev, inImg, image_size, hipMemcpyHostToDevice); checkErrors();
    guidedGpuShared(inImgDev, tmpImgDev, outImgDev, width, height, EPS);
    hipMemcpy(outImg, outImgDev, image_size, hipMemcpyDeviceToHost); checkErrors();
  });

  const auto timeWithoutCopy = measureTime("CUDA shared without copy", [inImgDev, tmpImgDev, outImgDev, width, height]() {
    guidedGpuShared(inImgDev, tmpImgDev, outImgDev, width, height, EPS);
    hipDeviceSynchronize(); checkErrors();
  });

  measureTime("CUDA global without copy", [inImgDev, tmpImgDev, outImgDev, width, height]() {
    guidedGpuGlobal(inImgDev, tmpImgDev, outImgDev, width, height, EPS);
    hipDeviceSynchronize(); checkErrors();
  });

  measureTime("CUDA texture without copy", [tmpA, tmpB, cuArrayA, cuArrayB, outImgDev, width, height]() {
    guidedGpuTexture(tmpA, tmpB, cuArrayA, cuArrayB, outImgDev, width, height, EPS);
    hipDeviceSynchronize(); checkErrors();
  });

  std::cout << "Copy time: " << (timeWithCopy - timeWithoutCopy) << " ms" << std::endl;

  SOIL_save_image(
    "gpu.tga",
    SOIL_SAVE_TYPE_TGA,
    width,
    height,
    3,
    outImg
  );

  hipFree(inImgDev); checkErrors();
  hipFree(outImgDev); checkErrors();

  return 0;
}
